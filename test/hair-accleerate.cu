#include "hip/hip_runtime.h"

#include "Common/Render.hpp"
#include <iostream>
#include <optional>

#include "Integrator/Integrator.hpp"
#include "Integrator/TraceHelper.h"
#include "SampleRecords/SurfaceScatterEvent.hpp"
#include "Common/ProgressReporter.h"
#include "Common/Parallel.h"
#include "Bsdfs/Reflection.hpp"
#include "Common/Debug.hpp"

#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/config.h>
#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/encodings/oneblob.h>
#include <concurrent_vector.h>

Json config = {
        {"loss",      {
                              {"otype", "RelativeL2"}
                      }},
        {"optimizer", {
                              {"otype", "Adam"},
                              // {"otype", "Shampoo"},
                              {"learning_rate", 1e-2},
                              {"beta1",           0.9f},
                              {"beta2",      0.99f},
                              {"l2_reg",            0.0f},
                              // The following parameters are only used when the optimizer is "Shampoo".
                              {"beta3", 0.9f},
                              {"beta_shampoo", 0.0f},
                              {"identity", 0.0001f},
                              {"cg_on_momentum", false},
                              {"frobenius_normalization", true},
                      }},
        {"encoding",  {
                              {"otype", "Composite"},

                              {"nested",        {
                                                        {
                                                                {"n_dims_to_encode", 3},
                                                                {"otype", "HashGrid"}
                                                        },
                                                        {
                                                                {"n_dims_to_encode", 3},
                                                                {"otype", "OneBlob"},
                                                                {"n_bins", 32},
                                                        },
                                                        {
                                                                {"n_dims_to_encode", 3},
                                                                {"otype", "OneBlob"},
                                                                {"n_bins", 32},
                                                        }
                                                }},
                      }},
        {"network",   {
                              {"otype", "FullyFusedMLP"},
                              // {"otype", "CutlassMLP"},
                              {"n_neurons",     64},
                              {"n_hidden_layers", 2},
                              {"activation", "ReLU"},
                              {"output_activation", "None"},
                      }},
};

using namespace tcnn;
using precision_t = network_precision_t;

__global__ void save(vec3 pos, vec3 dir, vec3 tangent, float *__restrict__ result) {
    result[0] = pos[0];
    result[1] = pos[1];
    result[2] = pos[2];
    result[3] = dir[0];
    result[4] = dir[1];
    result[5] = dir[2];
    result[6] = tangent[0];
    result[7] = tangent[1];
    result[8] = tangent[2];
}

__global__ void save_out(vec3 L, float *__restrict__ result) {
    result[0] = L[0];
    result[1] = L[1];
    result[2] = L[2];
}

class HairIntegrator : public PathIntegrator {
    const uint32_t batch_size = 1 << 18;
    const uint32_t n_input_dims = 9; //pos,tangent,dir
    const uint32_t n_output_dims = 3;// rgb color
    const int train_num = 256;
    hipStream_t training_stream;
    hipStream_t inference_stream;
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network;

//    GPUMatrix<float> training_target;
//    GPUMatrix<float> training_batch;
    std::atomic<int> train_count;

    std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer;
    int beta = 3;
public:
    HairIntegrator(std::shared_ptr<Camera> camera, std::shared_ptr<Sampler> sampler) : PathIntegrator(camera, sampler,
                                                                                                      Json()),
                                                                                       training_batch(n_input_dims,
                                                                                                      submit_train_batch),
                                                                                       training_target(n_output_dims,
                                                                                                       submit_train_batch),
                                                                                       predict_batch(n_input_dims,submit_prdict_batch),
                                                                                       predict_result(n_output_dims,submit_prdict_batch),
                                                                                       predict_pixel_vector(submit_prdict_batch),
                                                                                       host_predict_result(submit_prdict_batch * n_output_dims)
                                                                                                       {
        spdlog::info("Begin init NN");
        initNN();
        spdlog::info("End init NN");
        maxBounces = 100;
    }

    // void  trainTarget(const Scene & scene,int beta,ivec2 pos,Sampler * sampler ){
    //     while()
    //     auto ray  =  _camera->sampleRay(pos.x,pos.y,sampler->getNext2D());
    //     auto L = integrate(ray,scene,sampler,beta);
    //     auto LPrime = integrate(ray,sceene,sampler,std::numeric_limits<int>::max());
    //     auto E = L - LPrime;
    // }
    Ray getSampleRay(ivec2 res, Sampler *sampler) {
        vec2 samplePos = sampler->getNext2D();
        int x = samplePos.x * res.x;
        int y = samplePos.y * res.y;
        return _camera->sampleRay(x, y, sampler->getNext2D());
    }

    const int submit_train_batch = 128;
    const int submit_prdict_batch = 128 * 16;
    std::atomic<int> cur_submit_batch = 0;
    std::atomic<int> cur_predict_batch = 0;
    Concurrency::concurrent_vector<vec2> predict_pixel_vector;

    GPUMatrix<float> training_batch;
    GPUMatrix<float> training_target;

    GPUMatrix<float> predict_batch;
    GPUMatrix<float> predict_result;
    std::vector<float> host_predict_result;
    void trainNetWork(const Scene &scene, ivec2 res, Sampler *sampler) {
        auto ray = getSampleRay(res, sampler);
        vec3 pos, tangent, dir = ray.d, LPrime(0), L(0);

        ///reutrn pos,dir,tangent,LPrime
        while (true) {
            if (integrate(ray, scene, maxBounces, *sampler, pos, tangent, LPrime, L))
                break;
            ray = getSampleRay(res, sampler);
        }

        auto E = L - LPrime;
        cur_submit_batch++;
        save<<< 1, 1>>>(pos, dir, tangent, training_batch.data() + n_input_dims * cur_submit_batch);
        save_out<<<1, 1>>>(E, training_target.data() + n_output_dims * cur_submit_batch);
        if (cur_submit_batch % submit_train_batch == 0) {
            cur_submit_batch = 0;
            trainer->training_step(training_stream, training_batch, training_target);
        }
    }


    void save_predict() {
        cur_predict_batch = 0;
        network->inference(inference_stream, predict_batch, predict_result);
        CUDA_CHECK_THROW(hipMemcpy(host_predict_result.data(), predict_result.data(), host_predict_result.size(), hipMemcpyDeviceToHost));

        for (int i = 0; i < submit_prdict_batch; i++) {
            auto pixel = predict_pixel_vector[i];
            auto L = vec3(host_predict_result[3 * i], host_predict_result[3 * i + 1],
                          host_predict_result[3 * i + 2]);
            _camera->image->addPixel(pixel.x, pixel.y, L, true);
        }
       // predict_pixel_vector.clear();
    }

    void initNN() {
        // training_target=    GPUMatrix<float>(n_output_dims, batch_size);
        //  training_batch =    GPUMatrix<float>(n_input_dims, batch_size);

        //   hipStream_t inference_stream;
        CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
        training_stream = inference_stream;


        json encoding_opts = config.value("encoding", json::object());
        json loss_opts = config.value("loss", json::object());
        json optimizer_opts = config.value("optimizer", json::object());
        json network_opts = config.value("network", json::object());


        std::shared_ptr<Loss<precision_t>> loss{create_loss<precision_t>(loss_opts)};
        std::shared_ptr<Optimizer<precision_t>> optimizer{create_optimizer<precision_t>(optimizer_opts)};
        network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims, encoding_opts,
                                                                          network_opts);
        trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

    }

public:
    void render(const Scene &scene) override {
        auto tileSize = scene.options.tileSize;
        ivec2 renderBounds = _camera->image->resoulation();
        int width = _camera->image->width();
        int height = _camera->image->height();
        ivec2 numTiles{(renderBounds.x + tileSize - 1) / tileSize, (renderBounds.y + tileSize - 1) / tileSize};

        int num_threads = std::thread::hardware_concurrency();
        parallel_init(num_threads);

        int spp = scene.options.spp;
        int sppStep = scene.options.sppStep;


        ProgressReporter reporter(numTiles.x * numTiles.y);

        /// train one scenond

        for (int i = 0; i < spp; i++) {
            while (train_count++ < train_num)
                trainNetWork(scene, _camera->image->resoulation(), _sampler.get());
            parallel_for([&](const vec2 &tile) {
                int x0 = tile[0] * tileSize;
                int x1 = std::min(x0 + tileSize, width);
                int y0 = tile[1] * tileSize;
                int y1 = std::min
                        (y0 + tileSize, height);

                int seed = x0 * width + y0;
                std::unique_ptr<Sampler> tileSampler = _sampler->clone(seed);


                for (int y = y0; y < y1; y++) {
                    for (int x = x0; x < x1; x++) {
                        Ray ray = _camera->sampleRay(x, y, tileSampler->getNext2D());
                        vec3 pos, tangent, dir = ray.d, LPrime(0), L(0);
                        ///reutrn pos,dir,tangent,LPrime
                        bool hitHair = integrate(ray, scene, maxBounces, *tileSampler, pos, tangent, LPrime, L);
                        if (hitHair) {
                            save<<<1, 1>>>(pos, dir, tangent, predict_batch.data()+3*cur_predict_batch);
                            if(cur_predict_batch == submit_prdict_batch)
                                int k = 1;
                            predict_pixel_vector[cur_predict_batch] = ivec2(x,y);
                            if(++cur_predict_batch == submit_prdict_batch)
                               save_predict();
                        }
                        _camera->image->addPixel(x, y, L, true);
                    }
                    train_count = 0;
                }

            }, numTiles);
        }
        parallel_cleanup();
        _camera->image->save(scene.options.outputFileName, 1.f / spp);


    }

    bool integrate(const Ray &ray, const Scene &scene, int maxDepth, Sampler &sampler, vec3 &pos, vec3 &tangent,
                   vec3 &LPrime, vec3 &L) const {
        std::optional<Intersection> its;
        SurfaceEvent surfaceEvent;
        Spectrum thr(1.0);
        int bounces = minBounces;
        bool specularBounce = true;
        Ray _ray(ray);
        for (bounces = 0;; ++bounces) {
            if (bounces == beta)
                LPrime = L;
            its = scene.intersect(_ray);
            if (bounces == 0) {
                if (its.has_value()) {
                    tangent = its->tangent.value();
                    pos = its.value().p;
                }
            }
            if (specularBounce && bounces > minBounces) {
                if (its.has_value())
                    L += thr * its->Le(-_ray.d);
                else
                    for (auto light: scene.lights) {
                        if (light->flags == int(LightFlags::Infinite)) {
                            L += thr * light->Le(_ray);
                        }
                    }

            }

            if (!its.has_value() || bounces >= maxDepth)
                break;


            surfaceEvent = makeLocalScatterEvent(&its.value());
            if (its->bsdf->Pure(BSDF_FORWARD)) {
                _ray = surfaceEvent.sctterRay(_ray.d);
            } else {
                if (!its->bsdf->Pure(BSDF_PURE_SPECULR) && bounces < maxDepth - 1) {
                    Spectrum Ld = uniformSampleAllLights
                            (surfaceEvent, scene, sampler, nullptr);  //direct lighting
                    L += thr * Ld;
                }
                surfaceEvent.requestType = BSDF_ALL;
                Spectrum f = its->bsdf->sampleF(surfaceEvent, sampler.getNext2D(), false);
                if (isBlack(f) || surfaceEvent.pdf == 0)
                    break;
                BXDFType flags = surfaceEvent.sampleType;
                specularBounce = (flags & BSDF_SPECULAR) != 0;
                thr *= f / surfaceEvent.pdf;
                _ray = surfaceEvent.sctterRay();
                if (russian(bounces, sampler, thr))
                    break;
            }
        }
        if (bounces > 4) {
            int k = 1;
        }
        return bounces > 0;
    }

};


int main(int argc, const char *argv[]) {
    FileUtils::WorkingDir = argv[1];
    std::ifstream scene_file(FileUtils::WorkingDir + "scene.json");
    nlohmann::json j;
    scene_file >> j;
    scene_file.close();
    Render render(j);
   // render.integrator.reset(new HairIntegrator(render.camera, render.sampler));
    render.Go();
}


