#include "hip/hip_runtime.h"

#include "Common/Render.hpp"
#include <iostream>
#include <optional>

#include "Integrator/Integrator.hpp"
#include "Integrator/TraceHelper.h"
#include "SampleRecords/SurfaceScatterEvent.hpp"
#include "Common/ProgressReporter.h"
#include "Common/Parallel.h"
#include "Bsdfs/Reflection.hpp"
#include "Common/Debug.hpp"
#include "Texture/BitMapTexture.hpp"

#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/config.h>
#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/encodings/oneblob.h>
#include <concurrent_vector.h>

float img_width;
float img_height;
vec2 img_extent;

template<uint32_t stride>
__global__ void eval_image(uint32_t n_elements, hipTextureObject_t texture, float *__restrict__ xs_and_ys,
                           float *__restrict__ result) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements) return;

    uint32_t output_idx = i * stride;
    uint32_t input_idx = i * 2;

    float4 val = tex2D<float4>(texture, xs_and_ys[input_idx], xs_and_ys[input_idx + 1]);

    result[output_idx + 0] = val.x;
    result[output_idx + 1] = val.y;
    result[output_idx + 2] = val.z;

    for (uint32_t i = 3; i < stride; ++i) {
        result[output_idx + i] = 1;
    }
}

template<typename T>
__global__ void
to_ldr(const uint64_t num_elements, const uint32_t n_channels, const uint32_t stride, const T *__restrict__ in,
       uint8_t *__restrict__ out) {
    const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= num_elements) return;

    const uint64_t pixel = i / n_channels;
    const uint32_t channel = i - pixel * n_channels;

    out[i] = (uint8_t) (powf(fmaxf(fminf(in[pixel * stride + channel], 1.0f), 0.0f), 1.0f / 2.2f) * 255.0f + 0.5f);
}

template<typename T>
void
save_image(const T *image, int width, int height, int n_channels, int channel_stride, const std::string &filename) {
    tcnn::GPUMemory<uint8_t> image_ldr(width * height * n_channels);
    tcnn::linear_kernel(to_ldr<T>, 0, nullptr, width * height * n_channels, n_channels, channel_stride, image,
                        image_ldr.data());

    std::vector<uint8_t> image_ldr_host(width * height * n_channels);
    CUDA_CHECK_THROW(hipMemcpy(image_ldr_host.data(), image_ldr.data(), image_ldr.size(), hipMemcpyDeviceToHost));


    ImageIO::saveLdr(filename.c_str(), image_ldr_host.data(), width, height, n_channels, true);

}

using precision_t = tcnn::network_precision_t;

tcnn::GPUMemory<float> load_image(const std::string &filename, int &width, int &height) {
    // width * height * RGBA
    auto out = ImageIO::loadLdrNormalize(filename.c_str(), TexelConversion::REQUEST_RGB, width, height);
    // float* out = load_stbi(&width, &height, filename.c_str());

    tcnn::GPUMemory<float> result(width * height * 4);
    result.copy_from_host(out.get());
    //free(out); // release memory of image data

    return result;
}


void origin(BitMapTexture<vec3> *BitMaptexture, std::shared_ptr<tcnn::Trainer<float, precision_t, precision_t>> trainer,
            std::shared_ptr<tcnn::NetworkWithInputEncoding<precision_t>> network) {
    int width, height;


    Json config = {
            {"loss",      {
                                  {"otype", "RelativeL2"}
                          }},
            {"optimizer", {
                                  {"otype", "Adam"},
                                  // {"otype", "Shampoo"},
                                  {"learning_rate", 1e-2},
                                  {"beta1",           0.9f},
                                  {"beta2",      0.99f},
                                  {"l2_reg",            0.0f},
                                  // The following parameters are only used when the optimizer is "Shampoo".
                                  {"beta3", 0.9f},
                                  {"beta_shampoo", 0.0f},
                                  {"identity", 0.0001f},
                                  {"cg_on_momentum", false},
                                  {"frobenius_normalization", true},
                          }},
            {"encoding",  {
                                  {"otype", "OneBlob"},
                                  {"n_bins",        32},
                          }},
            {"network",   {
                                  {"otype", "FullyFusedMLP"},
                                  // {"otype", "CutlassMLP"},
                                  {"n_neurons",     64},
                                  {"n_hidden_layers", 4},
                                  {"activation", "ReLU"},
                                  {"output_activation", "None"},
                          }},
    };
    Json encoding_opts = config.value("encoding", Json::object());
    Json loss_opts = config.value("loss", Json::object());
    Json optimizer_opts = config.value("optimizer", Json::object());
    Json network_opts = config.value("network", Json::object());
    std::shared_ptr<tcnn::Loss<precision_t>> loss{tcnn::create_loss<precision_t>(loss_opts)};
    std::shared_ptr<tcnn::Optimizer<precision_t>> optimizer{tcnn::create_optimizer<precision_t>(optimizer_opts)};
    // network = std::make_shared<tcnn::NetworkWithInputEncoding<precision_t>>(2, 3, encoding_opts, network_opts);

    //  trainer = std::make_shared<tcnn::Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

    // Second step: create a cuda texture out of this image. It'll be used to generate training data efficiently on the fly
    tcnn::GPUMemory<float> image = load_image(
            "curly-hair_PT_GROUD_TROUTH.png", width, height);
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = image.data();
    resDesc.res.pitch2D.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    resDesc.res.pitch2D.width = width;
    resDesc.res.pitch2D.height = height;
    resDesc.res.pitch2D.pitchInBytes = width * 4 * sizeof(float);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.normalizedCoords = true;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;

    hipTextureObject_t texture;
    CUDA_CHECK_THROW(hipCreateTextureObject(&texture, &resDesc, &texDesc, nullptr));

    uint32_t n_coords = img_width * img_height;
    uint32_t n_coords_padded = tcnn::next_multiple(n_coords, tcnn::BATCH_SIZE_GRANULARITY);

    tcnn::GPUMemory<float> sampled_image(n_coords * 3);
    tcnn::GPUMemory<float> xs_and_ys(n_coords_padded * 2);

    std::vector<float> host_xs_and_ys(n_coords * 2);
    int sampling_height = img_width;
    int sampling_width = img_height;
    for (int y = 0; y < sampling_height; ++y) {
        for (int x = 0; x < sampling_width; ++x) {
            int idx = (y * sampling_width + x) * 2;
            host_xs_and_ys[idx + 0] = (float) (x + 0.5) / (float) sampling_width;
            host_xs_and_ys[idx + 1] = (float) (y + 0.5) / (float) sampling_height;
        }
    }

    xs_and_ys.copy_from_host(host_xs_and_ys.data());



    // Fourth step: train the model by sampling the above image and optimizing an error metric

    // Various constants for the network and optimization
    const uint32_t batch_size = 1 << 18;
    const uint32_t n_training_steps = 10000000;
    const uint32_t n_input_dims = 2; // 2-D image coordinate
    const uint32_t n_output_dims = 3; // RGB color

    hipStream_t inference_stream;
    CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
    hipStream_t training_stream = inference_stream;

    tcnn::pcg32 rng{1337};

    // Auxiliary matrices for training
    tcnn::GPUMatrix<float> training_target(n_output_dims, batch_size);
    tcnn::GPUMatrix<float> training_batch(n_input_dims, batch_size);

    // Auxiliary matrices for evaluation
    tcnn::GPUMatrix<float> prediction(n_output_dims, n_coords_padded);
    tcnn::GPUMatrix<float> inference_batch(xs_and_ys.data(), n_input_dims, n_coords_padded);


    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    float tmp_loss = 0;
    uint32_t tmp_loss_counter = 0;

    std::cout << "Beginning optimization with " << n_training_steps << " training steps." << std::endl;

    uint32_t interval = 10;

    for (uint32_t i = 0; i < 1000; ++i) {
        bool print_loss = i % interval == 0;
        bool visualize_learned_func = i % interval == 0;

        // Compute reference values at random coordinates
        {
            tcnn::generate_random_uniform<float>(training_stream, rng, batch_size * n_input_dims,
                                                 training_batch.data());
            tcnn::linear_kernel(eval_image<n_output_dims>, 0, training_stream, batch_size, texture,
                                training_batch.data(),
                                training_target.data());
        }

        // Training step
        {
            auto ctx = trainer->training_step(training_stream, training_batch, training_target);

            if (i % std::min(interval, (uint32_t) 100) == 0) {
                //tmp_loss += trainer->loss(training_stream, *ctx);
                ++tmp_loss_counter;
            }
        }

        // Debug outputs
        {
            if (print_loss) {
                std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
                std::cout << "Step#" << i << ": " << "loss=" << tmp_loss / (float) tmp_loss_counter << " time="
                          << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]"
                          << std::endl;

                tmp_loss = 0;
                tmp_loss_counter = 0;
            }
            if (visualize_learned_func) {
                network->inference(inference_stream, inference_batch, prediction);
                auto filename = fmt::format("{}.png", i);
                std::cout << "Writing '" << filename << "'... ";
                save_image(prediction.data(), sampling_width, sampling_height, 3, n_output_dims, filename);
                std::cout << "done." << std::endl;
            }
            // Don't count visualizing as part of timing
            // (assumes visualize_learned_pdf is only true when print_loss is true)
            if (print_loss) {
                begin = std::chrono::steady_clock::now();
            }
        }

        if (print_loss && i > 0 && interval < 1000) {
            interval *= 10;
        }
        tcnn::free_all_gpu_memory_arenas();

    }

    // Dump final image if a name was specified
}


Json config = {
        {"loss",      {
                              {"otype",  "RelativeL2"}
                      }},
        {"optimizer", {
                              {"otype",  "Adam"},
                              // {"otype", "Shampoo"},
                              {"learning_rate", 5e-3},
                              {"beta1",           0.9f},
                              {"beta2",      0.99f},
                              {"l2_reg",            0.0f},
                              // The following parameters are only used when the optimizer is "Shampoo".
                              {"beta3", 0.9f},
                              {"beta_shampoo", 0.0f},
                              {"identity", 0.0001f},
                              {"cg_on_momentum", false},
                              {"frobenius_normalization", true},
                      }},
        {"encoding",  {
                              {"nested", {
                                                 {
                                                         {"n_dims_to_encode", 3},
                                                         {"otype", "HashGrid "}
                                                 }, {
                                                            {"n_dims_to_encode", 3},
                                                            {"otype", "OneBlob"},
                                                            {"n_bins", 32},
                                                    },
                                                 {
                                                         {"n_dims_to_encode", 3},
                                                         {"otype", "OneBlob"},
                                                         {"n_bins", 32},
                                                 }

                                         }}
                      }},
        {"network",   {
                              {"otype",  "FullyFusedMLP"},
                              // {"otype", "CutlassMLP"},
                              {"n_neurons",     64},
                              {"n_hidden_layers", 2},
                              {"activation", "ReLU"},
                              {"output_activation", "None"},
                      }},
};

__global__ void save(vec2 uv, float *__restrict__ result) {
    result[0] = uv.x;
    result[1] = uv.y;

}

__global__ void save(vec3 pos, vec3 dir, vec3 tangent, float *__restrict__ result) {
    result[0] = pos[0];
    result[1] = pos[1];
    result[2] = pos[2];
    result[3] = dir[0];
    result[4] = dir[1];
    result[5] = dir[2];
    result[6] = tangent[0];
    result[7] = tangent[1];
    result[8] = tangent[2];
}

void cpu_save(vec3 pos, vec3 dir, vec3 tangent, float *result) {
    //  tangent = vec3(0);
    //spdlog::info(dir);
    result[0] = pos[0];
    result[1] = pos[1];
    result[2] = pos[2];
    result[3] = dir[0];
    result[4] = dir[1];
    result[5] = dir[2];
    result[6] = tangent[0];
    result[7] = tangent[1];
    result[8] = tangent[2];

//    result[0] = 0;
//    result[1] = 0;
//    result[2] = 0;
//    result[3] = 0;
//    result[4] = 0;
//    result[5] = 0;
//    result[6] = 0;
//    result[7] = 0;
//    result[8] = 0;

}

void help(hipStream_t training_stream, uint32_t batch_size, hipTextureObject_t texture,
          tcnn::GPUMatrix<float> &training_batch,
          tcnn::GPUMatrix<float> &training_target) {
    tcnn::linear_kernel(eval_image<3>, 0, training_stream, batch_size, texture,
                        training_batch.data(),
                        training_target.data());
}

__global__ void save_out(vec3 L, float *__restrict__ result) {
    result[0] = L[0];
    result[1] = L[1];
    result[2] = L[2];
}

__global__ void cuda_copy(uint32_t n_elements, float *__restrict__ src, float *__restrict__ dst) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements) return;
    uint32_t idx = 3 * i;
    dst[idx] = src[idx];
    dst[idx + 1] = src[idx + 1];
    dst[idx + 2] = src[idx + 2];
};

const Float uv_scale_factor = 1.f;

class HairIntegrator : public PathIntegrator {
    const uint32_t batch_size = 1 << 18;
    bool by_pos = true;
    bool by_dir = false;
    const uint32_t n_input_dims = 9; //pos,tangent,dir
    const uint32_t n_output_dims = 3;// rgb color
    const int train_num = 512 * 512;
    hipStream_t training_stream;
    hipStream_t inference_stream;

    std::shared_ptr<tcnn::NetworkWithInputEncoding<precision_t>> network;

//    GPUMatrix<float> training_target;
//    GPUMatrix<float> training_batch;
    std::atomic<int> train_count;

    std::shared_ptr<tcnn::Trainer<float, precision_t, precision_t>> trainer;
    int beta = 2;
public:
    HairIntegrator(std::shared_ptr<Camera> camera, std::shared_ptr<Sampler> sampler) : PathIntegrator(camera, sampler,
                                                                                                      Json()),
                                                                                       tangent_img(
                                                                                               camera->image->resoulation()),
                                                                                       pos_img(camera->image->resoulation()),
                                                                                       dir_img(camera->image->resoulation()),
                                                                                       hit_hair_img(
                                                                                               camera->image->resoulation()),
                                                                                       hit_hair(
                                                                                               camera->image->resoulation().x *
                                                                                               camera->image->resoulation().y,
                                                                                               false),
                                                                                       training_batch(n_input_dims,
                                                                                                      submit_train_batch),
                                                                                       training_target(n_output_dims,
                                                                                                       submit_train_batch),
                                                                                       predict_batch(n_input_dims,
                                                                                                     submit_prdict_batch),
                                                                                       predict_result(n_output_dims,
                                                                                                      submit_prdict_batch),
                                                                                       predict_pixel_vector(
                                                                                               submit_prdict_batch),
                                                                                       host_predict_result(
                                                                                               submit_prdict_batch *
                                                                                               n_output_dims),
                                                                                       host_predict_value(
                                                                                               submit_prdict_batch *
                                                                                               n_input_dims),
                                                                                       predict_value_vecotr(
                                                                                               submit_prdict_batch *
                                                                                               n_input_dims),
                                                                                       gt("") {
        if (n_input_dims == 2 || n_input_dims == 3) {
            config["encoding"] = {
                    {"otype", "HashGrid"}
            };
        }
        if (n_input_dims == 6) {
            config["encoding"] = {
                    {"nested", {
                            {
                                    {"n_dims_to_encode", 3},
                                    {"otype", "OneBlob"},
                                    {"n_bins", 32400},
                            },
                            {
                                    {"n_dims_to_encode", 3},
                                    {"otype", "OneBlob"},
                                    {"n_bins", 32400},
                            }

                    }}
            };
        }
        if (by_pos && n_input_dims == 3)
            config["encoding"] = {
                    {"otype", "HashGrid"}
            };
        spdlog::info("Begin init NN");
        initNN();
        spdlog::info("End init NN");
        maxBounces = 100;
    }

    // void  trainTarget(const Scene & scene,int beta,ivec2 pos,Sampler * sampler ){
    //     while()
    //     auto ray  =  _camera->sampleRay(pos.x,pos.y,sampler->getNext2D());
    //     auto L = integrate(ray,scene,sampler,beta);
    //     auto LPrime = integrate(ray,sceene,sampler,std::numeric_limits<int>::max());
    //     auto E = L - LPrime;
    // }

    Ray getSampleRay(ivec2 res, vec2 u1, vec2 u2) {
        int x = u1.x * res.x;
        int y = u1.y * res.y;
        return _camera->sampleRay(x, y, u2);
    }

    const int submit_train_batch = 256;
    const int submit_prdict_batch = submit_train_batch;
    std::atomic<int> cur_submit_batch = 0;
    std::atomic<int> cur_predict_batch = 0;
    Concurrency::concurrent_vector<vec2> predict_pixel_vector;

    tcnn::GPUMatrix<float> training_batch;
    tcnn::GPUMatrix<float> training_target;

    tcnn::GPUMatrix<float> predict_batch;
    tcnn::GPUMatrix<float> predict_result;
    std::vector<float> host_predict_result;
    std::vector<float> host_predict_value;
    std::vector<bool> hit_hair;
    Concurrency::concurrent_vector<float> predict_value_vecotr;
    BitMapTexture<vec3> gt;
    Image tangent_img, pos_img, dir_img, hit_hair_img;

//    void trainNetWork(const Scene &scene, ivec2 res, Sampler *sampler) {
//        auto ray = getSampleRay(res, sampler);
//        vec3 pos, tangent, dir(0), LPrime(0), L(0);
//
//        ///reutrn pos,dir,tangent,LPrime
//        while (true) {
//            if (integrate(ray, scene, maxBounces, *sampler, pos, dir, tangent, LPrime, L))
//                break;
//            ray = getSampleRay(res, sampler);
//        }
//
//        auto E = L - LPrime;
//        //E = vec3(0);
//        save<<< 1, 1>>>(pos, dir, tangent, training_batch.data() + n_input_dims * cur_submit_batch);
//        save_out<<<1, 1>>>(E, training_target.data() + n_output_dims * cur_submit_batch);
//        cur_submit_batch++;
//        if (cur_submit_batch % submit_train_batch == 0) {
//            cur_submit_batch = 0;
//            trainer->training_step(training_stream, training_batch, training_target);
//        }
//    }

    const int float_size_factor = 4;

    void save_predict() {
        ivec2 res = _camera->image->resoulation();
        std::vector<ivec2> pixel_pos;
        std::vector<float> predict_data;
        for (int i = 0; i < res.x; i++)
            for (int j = 0; j < res.y; j++) {
                if (!isBlack(hit_hair_img.getPixel(i + j * res.x))) {
                    pixel_pos.emplace_back(i, j);
                }
            }
        int hair_pixel_size = (pixel_pos.size() / tcnn::BATCH_SIZE_GRANULARITY) * tcnn::BATCH_SIZE_GRANULARITY;
        predict_data.resize(n_input_dims * hair_pixel_size);
        vec3 min_pos(1e5f);
        vec3 max_pos(-1e5f);
        for (int i = 0; i < hair_pixel_size; i++) {
            auto p = pixel_pos[i];
            int idx = p.x + p.y * res.x;
            if (n_input_dims == 9)
                cpu_save(pos_img.getPixel(idx), dir_img.getPixel(idx), tangent_img.getPixel(idx),
                         predict_data.data() + i * n_input_dims);
            if (n_input_dims == 2) {
                vec2 uv((p.x / img_width) * uv_scale_factor, (1 - p.y / img_height) * uv_scale_factor);
                if (by_pos)
                    uv = pos_img.getPixel(idx);
                (predict_data.data() + i * n_input_dims)[0] = uv.x;
                (predict_data.data() + i * n_input_dims)[1] = uv.y;

            }
            if (n_input_dims == 3) {
                auto vec = tangent_img.getPixel(idx);
                if (by_pos) {
                    vec = pos_img.getPixel(idx);
                    min_pos = min(vec, min_pos);
                    max_pos = max(vec, max_pos);
                }
                if (by_dir) vec = dir_img.getPixel(idx);
                (predict_data.data() + i * n_input_dims)[0] = vec[0];
                (predict_data.data() + i * n_input_dims)[1] = vec[1];
                (predict_data.data() + i * n_input_dims)[2] = vec[2];
            }
            if (n_input_dims == 6) {
                auto tangent = tangent_img.getPixel(idx);
                auto dir = dir_img.getPixel(idx);
                (predict_data.data() + i * n_input_dims)[0] = dir[0];
                (predict_data.data() + i * n_input_dims)[1] = dir[1];
                (predict_data.data() + i * n_input_dims)[2] = dir[2];
                dir = tangent;
                (predict_data.data() + i * n_input_dims)[3] = dir[0];
                (predict_data.data() + i * n_input_dims)[4] = dir[1];
                (predict_data.data() + i * n_input_dims)[5] = dir[2];
            }
        }


        predict_batch = tcnn::GPUMatrix<float>(n_input_dims, hair_pixel_size);
        predict_result = tcnn::GPUMatrix<float>(n_output_dims, hair_pixel_size);
        host_predict_result.resize(n_output_dims * hair_pixel_size);

        CUDA_CHECK_THROW(hipMemcpy(predict_batch.data(), predict_data.data(), predict_data.size() * float_size_factor,
                                    hipMemcpyHostToDevice));
        network->inference(inference_stream, predict_batch, predict_result);
        hipMemcpy(host_predict_result.data(), predict_result.data(), host_predict_result.size() * float_size_factor,
                   hipMemcpyDeviceToHost);
        // save_image(predict_result.data(), 1024, 1024, 3, n_output_dims, "predict.png");

        //   hit_hair_img.save("hit.png", 1,true);


        for (int i = 0; i < hair_pixel_size; i++) {
            auto pixel = pixel_pos[i];

            auto L = vec3(host_predict_result[3 * i], host_predict_result[3 * i + 1],
                          host_predict_result[3 * i + 2]);
            //    L= vec3(1);
            _camera->image->addPixel(pixel.x, pixel.y, L, false);
        }

    }


    void train_image(const Scene &scene, float &tmp_loss, bool count_loss) {

        // Debug outputs
        ivec2 res = _camera->image->resoulation();
        training_batch = tcnn::GPUMatrix<float>(n_input_dims, train_num);
        training_target = tcnn::GPUMatrix<float>(n_output_dims, train_num);
        std::vector<float> host_traing_batch(train_num * n_input_dims);
        std::vector<float> host_traing_target(train_num * n_output_dims);
        auto sampler = _sampler.get();
        vec3 min_pos(1e5f);
        vec3 max_pos(-1e5f);
        for (int i = 0; i < train_num; i++) {
            int x, y;
            vec2 u1 = sampler->getNext2D(), u2 = sampler->getNext2D();
            auto ray = getSampleRay(res, u1, u2);
            vec3 pos(0), tangent(0), dir(0), LPrime(0), L(0);
            while (true) {
                if (integrate(ray, scene, 1, *sampler, pos, dir, tangent, L)) {
                    //integrate(ray, scene, beta, *sampler, pos, dir, tangent, LPrime);
                    break;
                }
                u1 = sampler->getNext2D(), u2 = sampler->getNext2D();
                ray = getSampleRay(res, u1, u2);
            }
            min_pos = min(min_pos, pos);
            max_pos = max(max_pos, pos);
            vec3 E = (L - LPrime);
            auto uv = (u1 + u2 / img_extent ) * uv_scale_factor;
            if (n_input_dims == 2) {
                //       uv = sampler->getNext2D();
                E = gt.eval(uv);
            }
            E = gt.eval(uv);
             x = uv.x * img_width;
           y = uv.y * img_height;
            if (by_pos)
                uv = pos;

//            _camera->image->addPixel(x,y,E);
        //    pos_img.addPixel(x, y,vec3( uv.x,uv.y,0), true);


            if (n_input_dims == 9)
                cpu_save(pos, dir, tangent, host_traing_batch.data() + n_input_dims * i);
            if (n_input_dims == 2) {
                (host_traing_batch.data() + n_input_dims * i)[0] = uv.x;
                (host_traing_batch.data() + n_input_dims * i)[1] = uv.y;
            }
            if (n_input_dims == 3) {
                auto vec = tangent;
                if (by_pos) vec = pos;
                if (by_dir) vec = dir;
                (host_traing_batch.data() + i * n_input_dims)[0] = vec[0];
                (host_traing_batch.data() + i * n_input_dims)[1] = vec[1];
                (host_traing_batch.data() + i * n_input_dims)[2] = vec[2];
//                (host_traing_batch.data() + n_input_dims * i)[0] = uv.x;
//                (host_traing_batch.data() + n_input_dims * i)[1] = uv.y;
            }
            if (n_input_dims == 6) {
                (host_traing_batch.data() + i * n_input_dims)[0] = dir[0];
                (host_traing_batch.data() + i * n_input_dims)[1] = dir[1];
                (host_traing_batch.data() + i * n_input_dims)[2] = dir[2];
                dir = tangent;
                (host_traing_batch.data() + i * n_input_dims)[3] = dir[0];
                (host_traing_batch.data() + i * n_input_dims)[4] = dir[1];
                (host_traing_batch.data() + i * n_input_dims)[5] = dir[2];
            }
            host_traing_target[n_output_dims * i] = E[0];
            host_traing_target[n_output_dims * i + 1] = E[1];
            host_traing_target[n_output_dims * i + 2] = E[2];
        }
        if ((by_pos && (n_input_dims == 3 || n_input_dims == 2)) || n_input_dims == 9) {
            auto diff_pos = max_pos - min_pos;
            for (int i = 0; i < train_num; i++) {
                host_traing_batch[i * n_input_dims] =
                        (host_traing_batch[i * n_input_dims] - min_pos[0]) / (diff_pos[0]);
                host_traing_batch[i * n_input_dims + 1] =
                        (host_traing_batch[i * n_input_dims + 1] - min_pos[1]) / (diff_pos[1]);
//                host_traing_batch[i * n_input_dims] = uv.x;
//                host_traing_batch[i * n_input_dims + 1] = uv.y;
                if (n_input_dims >=3 )
                    host_traing_batch[i * n_input_dims + 2] =
                            (host_traing_batch[i * n_input_dims + 2] - min_pos[2]) / (diff_pos[2]);
            }
        }
      //  pos_img.linerarNormalize();
   //     pos_img.save("pos.exr", 1, false);
     //   exit(-1);

        CUDA_CHECK_THROW(hipMemcpy(training_batch.data(), host_traing_batch.data(),
                                    float_size_factor * host_traing_batch.size(), hipMemcpyHostToDevice));
        auto error = hipMemcpy(training_target.data(), host_traing_target.data(),
                                float_size_factor * host_traing_target.size(), hipMemcpyHostToDevice);

        int width = img_width;
        int height = img_height;
//        ImageIO::loadLdrNormalize("curly-hair_PT_GROUD_TROUTH.png", TexelConversion::REQUEST_RGB, width, height);
//        load_image(
//                "curly-hair_PT_GROUD_TROUTH.png", width, height);
//        tcnn::GPUMemory<float> image = load_image(
//                "curly-hair_PT_GROUD_TROUTH.png", width, height);
//        hipResourceDesc resDesc;
//        memset(&resDesc, 0, sizeof(resDesc));
//        resDesc.resType = hipResourceTypePitch2D;
//        resDesc.res.pitch2D.devPtr = image.data();
//        resDesc.res.pitch2D.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
//        resDesc.res.pitch2D.width = width;
//        resDesc.res.pitch2D.height = height;
//        resDesc.res.pitch2D.pitchInBytes = width * 4 * sizeof(float);
//
//        hipTextureDesc texDesc;
//        memset(&texDesc, 0, sizeof(texDesc));
//        texDesc.filterMode = hipFilterModeLinear;
//        texDesc.normalizedCoords = true;
//        texDesc.addressMode[0] = hipAddressModeClamp;
//        texDesc.addressMode[1] = hipAddressModeClamp;
//        texDesc.addressMode[2] = hipAddressModeClamp;

        //hipTextureObject_t texture;
        //CUDA_CHECK_THROW(hipCreateTextureObject(&texture, &resDesc, &texDesc, nullptr));
        tcnn::default_rng_t rng{1337};
//          tcnn::generate_random_uniform<float>(training_stream, rng, batch_size * n_input_dims, training_batch.data());
        //  help(training_stream, batch_size, texture, training_batch, training_target);
        std::vector<float> image_host_traing_target(host_traing_target.size(), 1);
        hipMemcpy(image_host_traing_target.data(), training_target.data(), host_traing_target.size() * 4,
                   hipMemcpyDeviceToHost);
        auto ctx = trainer->training_step(training_stream, training_batch, training_target);
        if (count_loss)
            tmp_loss += trainer->loss(training_stream, *ctx);

    }

    void initNN() {
        // training_target=    GPUMatrix<float>(n_output_dims, batch_size);
        //  training_batch =    GPUMatrix<float>(n_input_dims, batch_size);

        //   hipStream_t inference_stream;
        CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
        training_stream = inference_stream;


        Json encoding_opts = config.value("encoding", Json::object());
        Json loss_opts = config.value("loss", Json::object());
        Json optimizer_opts = config.value("optimizer", Json::object());
        Json network_opts = config.value("network", Json::object());


        std::shared_ptr<tcnn::Loss<precision_t>> loss{tcnn::create_loss<precision_t>(loss_opts)};
        std::shared_ptr<tcnn::Optimizer<precision_t>> optimizer{tcnn::create_optimizer<precision_t>(optimizer_opts)};
        network = std::make_shared<tcnn::NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims,
                                                                                encoding_opts,
                                                                                network_opts);

        trainer = std::make_shared<tcnn::Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

    }

public:
    void render(const Scene &scene) override {
        // origin(nullptr, trainer, network);
        //  return;
        gt = BitMapTexture<vec3>("curly-hair_PT_GROUD_TROUTH.png");
        gt.LoadResources();
        auto tileSize = scene.options.tileSize;
        ivec2 renderBounds = _camera->image->resoulation();
        int width = _camera->image->width();
        int height = _camera->image->height();
        ivec2 numTiles{(renderBounds.x + tileSize - 1) / tileSize, (renderBounds.y + tileSize - 1) / tileSize};

        int num_threads = std::thread::hardware_concurrency();
        parallel_init(num_threads);

        int spp = scene.options.spp;
        spp = 1000;
        int sppStep = scene.options.sppStep;


        ProgressReporter reporter(numTiles.x * numTiles.y);

        std::vector<std::unique_ptr<Sampler>> samplers(numTiles.x * numTiles.y);
        for (int x = 0; x < numTiles.x; x++)
            for (int y = 0; y < numTiles.y; y++) {
                int seed = y * tileSize + x;
                samplers[seed] = std::move(_sampler->clone(seed));
            }
        /// train one scenond

        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

        float tmp_loss = 0;
        for (int i = 0; i < spp; i++) {
            int interval = 10;
            bool print_loss = i % interval == 0;
            print_loss = true;
            train_image(scene, tmp_loss, print_loss);

            if (print_loss) {
                std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
                std::cout << "Step#" << i << ": " << "loss=" << tmp_loss << " time="
                          << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]"
                          << std::endl;
                tmp_loss = 0;
            }
            parallel_for([&](const vec2 &tile) {
                //  return ;
                int x0 = tile[0] * tileSize;
                int x1 = std::min(x0 + tileSize, width);
                int y0 = tile[1] * tileSize;
                int y1 = std::min
                        (y0 + tileSize, height);
                auto tileSampler = samplers[tile.y * tileSize + tile.x].get();
                for (int y = y0; y < y1; y++) {
                    for (int x = x0; x < x1; x++) {
                        Ray ray = _camera->sampleRay(x, y, tileSampler->getNext2D());
                        vec3 pos(0), tangent(0), dir(0), LPrime(0), L(0);
                        ///reutrn pos,dir,tangent,LPrime
//                           Spectrum  l = PathIntegrator::integrate(ray,scene,*tileSampler);
//                            _camera->image->addPixel(x, y, l, true);
//                             continue;
                        bool hitHair = true;
                        if (n_input_dims == 9 || n_input_dims == 3 || n_input_dims == 6) {
                            hitHair = integrate(ray, scene, beta, *tileSampler, pos, dir, tangent, L);
                        }
                        if(n_input_dims ==2 )
                            hitHair = integrate(ray, scene,1, *tileSampler, pos, dir, tangent, L);
                        if (hitHair) {
                            //integrate(ray, scene,maxBounces, *tileSampler, pos, dir, tangent, L);
                            tangent_img.addPixel(x, y, tangent, true);
                            if(hasNan(pos)){
                                int k = 1;
                            }
                            pos_img.addPixel(x, y, pos);
                            dir_img.addPixel(x, y, dir);
                            hit_hair_img.addPixel(x, y, Spectrum(1));
                            _camera->image->addPixel(x, y, vec3(0), true);
                        } else _camera->image->addPixel(x, y, L, true);

                    }
                    train_count = 0;
                }

            }, numTiles);
            //  tangent_img.save("curly-hair-tangent.png", 1.f, true);
            auto getFileName = [](std::string name, int i) {
                return name + std::to_string(i) + ".png";
            };
//              tangent_img.save(getFileName("tangent",i) ,1, true);
//                dir_img.save(getFileName("dir",i),1, true);
//              pos_img.save(getFileName("pos",i), 1,true);
//              exit(-1);
            //      pos_img.normalize();
//           dir_img.normalize();
//            tangent_img.normalize();
            pos_img.linerarNormalize();
            save_predict();
            tcnn::free_all_gpu_memory_arenas();
            _camera->image->save(std::to_string(i) + ".png", 1, true);


            pos_img.clear();
            dir_img.clear();
            hit_hair_img.clear();
            pos_img.clear();


        }
        parallel_cleanup();
        _camera->image->save(scene.options.outputFileName, 1.f / spp);

    }

    bool
    integrate(const Ray &ray, const Scene &scene, int maxDepth, Sampler &sampler, vec3 &pos, vec3 &dir, vec3 &tangent,
              vec3 &L) const {
        std::optional<Intersection> its;
        SurfaceEvent surfaceEvent;
        Spectrum thr(1.0);
        bool specularBounce = true;
        Ray _ray(ray);
        int bounces = 0;
        for (bounces = 0;; ++bounces) {

            its = scene.intersect(_ray);

            if (specularBounce && bounces > minBounces) {
                if (its.has_value())
                    L += thr * its->Le(-_ray.d);
                else
                    for (auto light: scene.lights) {
                        if (light->flags == int(LightFlags::Infinite)) {
                            L += thr * light->Le(_ray);
                        }
                    }

            }
//            if (bounces == beta)
//                LPrime = L;
            if (!its.has_value() || bounces >= maxDepth)

                break;


            surfaceEvent = makeLocalScatterEvent(&its.value());
            if (bounces == beta) {
                if (its.has_value()) {
                    tangent = its->tangent.value();
                      tangent = its->Ng;
                    pos = its.value().p;
                    dir = surfaceEvent.wo;
                    dir = ray.d;
                  //  dir = its->Ng;
                }
            }
            if (its->bsdf->Pure(BSDF_FORWARD)) {
                _ray = surfaceEvent.sctterRay(_ray.d);
            } else {
                if (!its->bsdf->Pure(BSDF_PURE_SPECULR) && bounces < maxDepth - 1) {
                    Spectrum Ld = uniformSampleAllLights
                            (surfaceEvent, scene, sampler, nullptr);  //direct lighting
                    L += thr * Ld;
                }
                surfaceEvent.requestType = BSDF_ALL;
                Spectrum f = its->bsdf->sampleF(surfaceEvent, sampler.getNext2D(), false);
                if (isBlack(f) || surfaceEvent.pdf == 0)
                    break;
                BXDFType flags = surfaceEvent.sampleType;
                specularBounce = (flags & BSDF_SPECULAR) != 0;
                thr *= f / surfaceEvent.pdf;
                _ray = surfaceEvent.sctterRay();
                // if(bounces==1) dir = _ray.d;
                if (russian(bounces, sampler, thr))
                    break;
            }
        }
        if (bounces > 4) {
            int k = 1;
        }
//        if (bounces < beta)
//            LPrime = L;
        //   L = vec3(bounces/10.f);
        return bounces > 0 ;
    }

};


int main(int argc, const char *argv[]) {
    img_width = 300;
    img_height = 1000;
    img_extent = vec2(img_width,img_height);
    FileUtils::WorkingDir = argv[1];
    std::ifstream scene_file(FileUtils::WorkingDir + "scene.json");
    nlohmann::json j;
    scene_file >> j;
    scene_file.close();
    Render render(j);
    render.integrator.reset(new HairIntegrator(render.camera, render.sampler));
    render.Go();
}


//using json = Json;
//int main(int argc, char* argv[]) {
//    try {
//        uint32_t compute_capability = tcnn::cuda_compute_capability();
//        if (compute_capability < tcnn::MIN_GPU_ARCH) {
//            std::cerr
//                    << "Warning: Insufficient compute capability " << compute_capability << " detected. "
//                    << "This program was compiled for >=" << tcnn::MIN_GPU_ARCH << " and may thus behave unexpectedly." << std::endl;
//        }
//
//        if (argc < 2) {
//            std::cout << "USAGE: " << argv[0] << " " << "path-to-image.jpg [path-to-optional-config.json]" << std::endl;
//            std::cout << "Sample EXR files are provided in 'data/images'." << std::endl;
//            return 0;
//        }
//
//        json config = {
//                {"loss", {
//                                 {"otype", "RelativeL2"}
//                         }},
//                {"optimizer", {
//                                 {"otype", "Adam"},
//                                 // {"otype", "Shampoo"},
//                                 {"learning_rate", 1e-2},
//                                 {"beta1", 0.9f},
//                                 {"beta2", 0.99f},
//                                 {"l2_reg", 0.0f},
//                                 // The following parameters are only used when the optimizer is "Shampoo".
//                                 {"beta3", 0.9f},
//                                 {"beta_shampoo", 0.0f},
//                                 {"identity", 0.0001f},
//                                 {"cg_on_momentum", false},
//                                 {"frobenius_normalization", true},
//                         }},
//                {"encoding", {
//                                 {"otype", "OneBlob"},
//                                 {"n_bins", 32},
//                         }},
//                {"network", {
//                                 {"otype", "FullyFusedMLP"},
//                                 // {"otype", "CutlassMLP"},
//                                 {"n_neurons", 64},
//                                 {"n_hidden_layers", 4},
//                                 {"activation", "ReLU"},
//                                 {"output_activation", "None"},
//                         }},
//        };
//
//        if (argc >= 3) {
//            std::cout << "Loading custom json config '" << argv[2] << "'." << std::endl;
//            std::ifstream f{argv[2]};
//            config = json::parse(f, nullptr, true, /*skip_comments=*/true);
//        }
//
//        // First step: load an image that we'd like to learn
//        int width, height;
//        tcnn::GPUMemory<float> image = load_image(argv[1], width, height);
//
//        // Second step: create a cuda texture out of this image. It'll be used to generate training data efficiently on the fly
//        hipResourceDesc resDesc;
//        memset(&resDesc, 0, sizeof(resDesc));
//        resDesc.resType = hipResourceTypePitch2D;
//        resDesc.res.pitch2D.devPtr = image.data();
//        resDesc.res.pitch2D.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
//        resDesc.res.pitch2D.width = width;
//        resDesc.res.pitch2D.height = height;
//        resDesc.res.pitch2D.pitchInBytes = width * 4 * sizeof(float);
//
//        hipTextureDesc texDesc;
//        memset(&texDesc, 0, sizeof(texDesc));
//        texDesc.filterMode = hipFilterModeLinear;
//        texDesc.normalizedCoords = true;
//        texDesc.addressMode[0] = hipAddressModeClamp;
//        texDesc.addressMode[1] = hipAddressModeClamp;
//        texDesc.addressMode[2] = hipAddressModeClamp;
//
//        hipTextureObject_t texture;
//        CUDA_CHECK_THROW(hipCreateTextureObject(&texture, &resDesc, &texDesc, nullptr));
//
//        // Third step: sample a reference image to dump to disk. Visual comparison of this reference image and the learned
//        //             function will be eventually possible.
//
//        int sampling_width = width;
//        int sampling_height = height;
//
//        // Uncomment to fix the resolution of the training task independent of input image
//        // int sampling_width = 1024;
//        // int sampling_height = 1024;
//
//        uint32_t n_coords = sampling_width * sampling_height;
//        uint32_t n_coords_padded = tcnn::next_multiple(n_coords, tcnn::batch_size_granularity);
//
//        tcnn::GPUMemory<float> sampled_image(n_coords * 3);
//        tcnn::GPUMemory<float> xs_and_ys(n_coords_padded * 2);
//
//        std::vector<float> host_xs_and_ys(n_coords * 2);
//        for (int y = 0; y < sampling_height; ++y) {
//            for (int x = 0; x < sampling_width; ++x) {
//                int idx = (y * sampling_width + x) * 2;
//                host_xs_and_ys[idx+0] = (float)(x + 0.5) / (float)sampling_width;
//                host_xs_and_ys[idx+1] = (float)(y + 0.5) / (float)sampling_height;
//            }
//        }
//
//        xs_and_ys.copy_from_host(host_xs_and_ys.data());
//
//        tcnn::linear_kernel(eval_image<3>, 0, nullptr, n_coords, texture, xs_and_ys.data(), sampled_image.data());
//
//        save_image(sampled_image.data(), sampling_width, sampling_height, 3, 3, "reference.jpg");
//
//        // Fourth step: train the model by sampling the above image and optimizing an error metric
//
//        // Various constants for the network and optimization
//        const uint32_t batch_size = 1 << 18;
//        const uint32_t n_training_steps = argc >= 4 ? atoi(argv[3]) : 10000000;
//        const uint32_t n_input_dims = 2; // 2-D image coordinate
//        const uint32_t n_output_dims = 3; // RGB color
//
//        hipStream_t inference_stream;
//        CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
//        hipStream_t training_stream = inference_stream;
//
//        tcnn::default_rng_t rng{1337};
//
//        // Auxiliary matrices for training
//        tcnn::GPUMatrix<float> training_target(n_output_dims, batch_size);
//        tcnn::GPUMatrix<float> training_batch(n_input_dims, batch_size);
//
//        // Auxiliary matrices for evaluation
//        tcnn::GPUMatrix<float> prediction(n_output_dims, n_coords_padded);
//        tcnn::GPUMatrix<float> inference_batch(xs_and_ys.data(), n_input_dims, n_coords_padded);
//
//        json encoding_opts = config.value("encoding", json::object());
//        json loss_opts = config.value("loss", json::object());
//        json optimizer_opts = config.value("optimizer", json::object());
//        json network_opts = config.value("network", json::object());
//
//        std::shared_ptr<tcnn::Loss<precision_t>> loss{tcnn::create_loss<precision_t>(loss_opts)};
//        std::shared_ptr<tcnn::Optimizer<precision_t>> optimizer{tcnn::create_optimizer<precision_t>(optimizer_opts)};
//        std::shared_ptr<tcnn::NetworkWithInputEncoding<precision_t>> network = std::make_shared<tcnn::NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims, encoding_opts, network_opts);
//
//        auto trainer = std::make_shared<tcnn::Trainer<float, precision_t, precision_t>>(network, optimizer, loss);
//
//        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
//
//        float tmp_loss = 0;
//        uint32_t tmp_loss_counter = 0;
//
//        std::cout << "Beginning optimization with " << n_training_steps << " training steps." << std::endl;
//
//        uint32_t interval = 10;
//
//        for (uint32_t i = 0; i < n_training_steps; ++i) {
//            bool print_loss = i % interval == 0;
//            bool visualize_learned_func = argc < 5 && i % interval == 0;
//
//            // Compute reference values at random coordinates
//            {
//                tcnn::generate_random_uniform<float>(training_stream, rng, batch_size * n_input_dims, training_batch.data());
//                tcnn::linear_kernel(eval_image<n_output_dims>, 0, training_stream, batch_size, texture, training_batch.data(), training_target.data());
//            }
//
//            // Training step
//            {
//                auto ctx = trainer->training_step(training_stream, training_batch, training_target);
//
//                if (i % std::min(interval, (uint32_t)100) == 0) {
//                    tmp_loss += trainer->loss(training_stream, *ctx);
//                    ++tmp_loss_counter;
//                }
//            }
//
//            // Debug outputs
//            {
//                if (print_loss) {
//                    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
//                    std::cout << "Step#" << i << ": " << "loss=" << tmp_loss/(float)tmp_loss_counter << " time=" << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
//
//                    tmp_loss = 0;
//                    tmp_loss_counter = 0;
//                }
//
//                if (visualize_learned_func) {
//                    network->inference(inference_stream, inference_batch, prediction);
//                    auto filename = fmt::format("{}.png", i);
//                    std::cout << "Writing '" << filename << "'... ";
//                    save_image(prediction.data(), sampling_width, sampling_height, 3, n_output_dims, filename);
//                    std::cout << "done." << std::endl;
//                }
//
//                // Don't count visualizing as part of timing
//                // (assumes visualize_learned_pdf is only true when print_loss is true)
//                if (print_loss) {
//                    begin = std::chrono::steady_clock::now();
//                }
//            }
//
//            if (print_loss && i > 0 && interval < 1000) {
//                interval *= 10;
//            }
//        }
//
//        // Dump final image if a name was specified
//        if (argc >= 5) {
//            network->inference(inference_stream, inference_batch, prediction);
//            save_image(prediction.data(), sampling_width, sampling_height, 3, n_output_dims, argv[4]);
//        }
//
//        tcnn::free_all_gpu_memory_arenas();
//
//        // If only the memory arenas pertaining to a single stream are to be freed, use
//        //free_gpu_memory_arena(stream);
//    } catch (std::exception& e) {
//        std::cout << "Uncaught exception: " << e.what() << std::endl;
//    }
//
//    return EXIT_SUCCESS;
//}
//
//
